#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <set>
using namespace std;

void run(int N)
{
    // int N = atoi(argv[1]);

    // Matrix dimensions
    int m = N; // Number of rows in A and C
    int n = N; // Number of columns in B and C
    int k = N; // Number of columns in A and rows in B

    // Allocate memory for matrices (row-major order)
    double *h_A = (double *)malloc(m * k * sizeof(double));
    double *h_B = (double *)malloc(k * n * sizeof(double));
    // double *h_C = (double *)malloc(m * n * sizeof(double));

    srand48(0); // seed

    // Initialize A and B with some values
    for (int i = 0; i < m * k; i++) h_A[i] = drand48();
    for (int i = 0; i < k * n; i++) h_B[i] = drand48();

    // Device matrices
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    // Copy matrices A, B, and C to the device
    hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);
    // cudaMemcpy(d_C, h_C, m * n * sizeof(double), cudaMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Set alpha and beta
    double alpha = 1.0;
    double beta = 0.0;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record the start event
    hipEventRecord(start, 0);

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n, k,
                &alpha,
                d_A, m,
                d_B, k,
                &beta,
                d_C, m);


    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // Wait for the GPU to finish before exiting
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Runtime Error after kernel execution: %s\n", hipGetErrorString(err));
    }

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    // printf("Matrix multiplication completed in %.6f seconds\n", elapsed_time_ms/1000);

    // Copy result back to host
    // cudaMemcpy(h_C, d_C, m * n * sizeof(double), cudaMemcpyDeviceToHost);

    // Print result
    // printf("Matrix C (result):\n");
    // for (int i = 0; i < m; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%8.2f ", h_C[i * n + j]);
    //     }
    //     printf("\n");
    // }
    printf("%d,%0.6f\n", N, elapsed_time_ms);
    fflush(stdout);

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
}

int main(int argc, char *argv[]) {
    int n = 1024;
    run(n);
    return 0;
}
